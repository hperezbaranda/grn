#include "hip/hip_runtime.h"
#include "common.h"
#include <iostream>
#include <stdio.h>
#include <cstdlib>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define SIZE 3

/*
* A simple introduction to programming in CUDA. This program prints "Hello
* World from GPU! from 10 CUDA threads running on the GPU.
*/
using namespace std;


void printData(const char *msg, int *in,  const int size)
{
    printf("%s: ", msg);

    for (int i = 0; i < size; i++)
    {
        printf("%5d", in[i]);
        fflush(stdout);
    }

    printf("\n");
    return;
}

__device__ bool state[SIZE];
//__shared__ bool equ[SIZE];

__device__ int getDecValue(bool* v){
  int num=0;
  for (size_t i = 0; i < SIZE; i++) {
    //printf("%sd ", (1 << i-1) * v[i-1] );
    num += (1 << i) * v[SIZE-i-1];
  }
  return num;
}

__device__ void initialState(unsigned long int valor, bool *vet, int size) {
  // std::cout <<" initialState: "<< std::endl;
  for (unsigned int i = 0; i < size; i++) {
    vet[size-1-i] = (valor & 1) != 0;
    valor >>= 1;
  }
}

__device__ void calculateState(bool* state,int num){
  // for (size_t i = 0; i < SIZE; i++) {
  //   printf("\n%d\n", equ[i]);
  // }
  // printf("recive: %d\n", num);
  bool equ[SIZE];
  equ[0] = ( state[0]  | !  state[2] );
  equ[1] = ( state[0]  &   state[2] );
  equ[2] = state[1];

  // bool change = true;
  // // if (c_fix > 0){
  //   for (size_t i = 0; i < c_fix; i++) {
  //     if(num == fix[i]){
  //       change = false;
  //     }
  //   }
  // }
  // if(change){
  //   1+1;
  // }
    state[num] = equ[num];

}

__global__ void initSharedMem(int c_fix, float *d_fix){
  for (unsigned int i = 0; i < c_fix+1; i++) {
    printf("%f\n",d_fix[i] );
  }
}

__global__ void findAttractor()
{

  bool state[SIZE];
  uint idx = blockDim.x * blockIdx.x + threadIdx.x;
  initialState(idx,state,SIZE);

  printf("thread number: %d\n", idx);
  // printf("My number: %d\n", number);
  hiprandState_t a_number;
  hiprand_init(idx+clock(), 0, 3,  &a_number);
  unsigned int aleatory = hiprand(&a_number);
  printf("aleatory %d\n", aleatory%SIZE );


  calculateState(state,aleatory%SIZE);
  __syncthreads();
// //  printf("saida: " );
  printf("%d\n", getDecValue(state));
}

__global__ void setRowReadRow(int *out)
{
    // static shared memory
    __shared__ int tile[3][3];

    // mapping from thread index to global memory index
    // unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;;

    // shared memory store operation
    tile[threadIdx.y][threadIdx.x] = idx;

    // wait for all threads to complete
    // __syncthreads();

    // shared memory load operation
    out[idx] = tile[threadIdx.y][threadIdx.x];
}

int main(int argc, char **argv)
{
  uint seed = (uint) time(NULL);
  srand(seed);
  uint suffle = rand() %SIZE;
  // std::cout <<   << std::endl;
  size_t numSimu = 1 << 0;
  size_t numState = 1 << 3;
  size_t numBlock =1;
  size_t numThreads =1;
  if (numState > 1024){
    numBlock = numState/1024;
    numThreads = 1024;
  }else{
    numThreads = numState;
  }

  for (size_t i = 0; i < numSimu; i++) {
    findAttractor<<<numBlock, numThreads>>>();
  }
  CHECK(hipDeviceReset());
  // hipDeviceReset();

  return 0;
}

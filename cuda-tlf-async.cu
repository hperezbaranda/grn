#include "hip/hip_runtime.h"
#include "common.h"
#include <iostream>
#include <stdio.h>
#include <cstdlib>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define SIZE 3

/*
 * A simple introduction to programming in CUDA. This program prints "Hello
 * World from GPU! from 10 CUDA threads running on the GPU.
 */
using namespace std;

__device__ int getDecValue(bool* v){
  int num=0;
  for (size_t i = 0; i < SIZE; i++) {
    //printf("%sd ", (1 << i-1) * v[i-1] );
    num += (1 << i) * v[SIZE-i-1];
  }
    return num;
}
__device__ void initialState(unsigned long int valor, bool *vet, int size) {
    // std::cout <<" initialState: "<< std::endl;
    for (int i = 0; i < size; i++) {
        vet[size-1-i] = (valor & 1) != 0;
        valor >>= 1;
        // printf("%d     ",vet[(size-i)-1] );
            // cout << vet[i] << " ";
    }
    // __syncthreads();
    // cout << endl;
}

__device__ void calculateState(bool* vet, int num){
  bool aux[SIZE]={  ( vet[0] ) | ! ( vet[2] )  , ( vet[0] ) &  ( vet[2] ) ,vet[1]};

  vet[num] = aux[num];
}

__global__ void findAttractor(int number)
{

    bool grafo[SIZE];
    uint thread = blockDim.x * blockIdx.x + threadIdx.x;
    initialState(thread,grafo,SIZE);
    // printf("%d\n", SIZE );

    printf("Hello World from GPU! number: %d\n", thread);
    // for (size_t i = 0; i < SIZE; i++) {
    //   printf("%d ",grafo[i] );
    // }
    //
    // printf("\n" );


    // hiprandState state;
    // uint seed = (uint) clock64();
    // hiprand_init(seed+thread,0,1,&state);
    // printf("State: %d\n", state);
    printf("My number: %d\n", number);
    calculateState(grafo,number);

    for (size_t i = 0; i < SIZE; i++) {
      printf("%d ",grafo[i] );
    }
    printf("\n%d", getDecValue(grafo));


}

int main(int argc, char **argv)
{
    printf("Hello World from CPU!\n");
    uint seed = (uint) time(NULL);
    srand(seed);
    uint suffle = rand() %SIZE;
    // std::cout <<   << std::endl;
    size_t numThreads =8;
    findAttractor<<<1, numThreads>>>(suffle);
    cout << endl;
    CHECK(hipDeviceReset());
    // hipDeviceReset();
    return 0;
}

#include "hip/hip_runtime.h"
#include "common.h"
#include <iostream>
#include <stdio.h>
#include <cstdlib>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define SIZE 3

/*
* A simple introduction to programming in CUDA. This program prints "Hello
* World from GPU! from 10 CUDA threads running on the GPU.
*/
using namespace std;


void printData(const char *msg, int *in,  const int size)
{
    printf("%s: ", msg);

    for (int i = 0; i < size; i++)
    {
        printf("%5d", in[i]);
        fflush(stdout);
    }

    printf("\n");
    return;
}

__device__ bool state[SIZE];
//__shared__ bool equ[SIZE];

__device__ int getDecValue(bool* v){
  int num=0;
  for (size_t i = 0; i < SIZE; i++) {
    //printf("%sd ", (1 << i-1) * v[i-1] );
    num += (1 << i) * v[SIZE-i-1];
  }
  return num;
}

__device__ void initialState(unsigned long int valor, bool *vet, int size) {
  // std::cout <<" initialState: "<< std::endl;
  for (unsigned int i = 0; i < size; i++) {
    vet[size-1-i] = (valor & 1) != 0;
    valor >>= 1;
  }
}

__device__ void calculateState(bool* state,int num){

  bool equ[SIZE];
  equ[0] = ( state[0]  | !  state[2] );
  equ[1] = ( state[0]  &   state[2] );
  equ[2] = state[1];

    state[num] = equ[num];

}

__global__ void findAttractor(int num, int * fix)
{

  bool state[SIZE];
  uint idx = blockDim.x * blockIdx.x + threadIdx.x;
  initialState(idx,state,SIZE);

  printf("thread number: %d\n", idx);
  // printf("My number: %d\n", number);
  hiprandState_t a_number;
  hiprand_init(idx+clock(), 0, 3,  &a_number);
  unsigned int aleatory = hiprand(&a_number);
  printf("aleatory %d\n", aleatory%SIZE );


  calculateState(state,aleatory%SIZE);
  __syncthreads();
  //  printf("saida: " );
  printf("%d\n", getDecValue(state));
}
__global__ void findAttractor(){

  bool state[SIZE];
  uint idx = blockDim.x * blockIdx.x + threadIdx.x;
  initialState(idx,state,SIZE);

  printf("thread number: %d\n", idx);
  // printf("My number: %d\n", number);
  hiprandState_t a_number;
  hiprand_init(idx+clock(), 0, 3,  &a_number);
  unsigned int aleatory = hiprand(&a_number);
  printf("aleatory %d\n", aleatory%SIZE );


  calculateState(state,aleatory%SIZE);
  __syncthreads();
  //  printf("saida: " );
  printf("%d\n", getDecValue(state));
}

__global__ void setRowReadRow(int *out){
    // static shared memory
    __shared__ int tile[3][3];

    // mapping from thread index to global memory index
    // unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // shared memory store operation
    tile[threadIdx.y][threadIdx.x] = idx;
    printf("%d\n", tile[threadIdx.y][threadIdx.x]);

    // wait for all threads to complete
    // __syncthreads();

    // shared memory load operation
    out[idx] = tile[threadIdx.y][threadIdx.x];
}

__global__ void testeNum(int num, int * fix){
  for (size_t i = 0; i < num; i++) {
    printf("%d\n",fix[i] );
  }
}

__global__ void testeNum1(){

    printf("%d\n",4 );

}

int main(int argc, char **argv)
{
  uint seed = (uint) time(NULL);
  srand(seed);
  uint suffle = rand() %SIZE;
  // std::cout <<   << std::endl;
  size_t numSimu = 1 << 0;
  size_t numState = 1 << 3;
  size_t numBlock =1;
  size_t numThreads =1;
  if (numState > 1024){
    numBlock = numState/1024;
    numThreads = 1024;
  }else{
    numThreads = numState;
  }

  // for (size_t i = 0; i < numSimu; i++) {
  //   findAttractor<<<numBlock, numThreads>>>();
  // }
  int numFix=0;
  if(argc > 1){
    numFix= atoi(argv[1]);
    int nBytes = numFix * sizeof(int);
    int *d_C;
    CHECK(hipMalloc((int**)&d_C, nBytes));
    int *gpuRef  = (int *)malloc(nBytes);
    int count =2;
    for (size_t i = 0; i < numFix; i++) {
      gpuRef[i] =atoi(argv[count++]);
    }
    CHECK(hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice));
    // testeNum<<<1, 3>>>(numFix,d_C);
    for (size_t i = 0; i < numSimu; i++) {
      findAttractor<<<numBlock, numThreads>>>(numFix,d_C);
    }
  }else{
    // testeNum1<<<1, 3>>>();
    for (size_t i = 0; i < numSimu; i++) {
      findAttractor<<<numBlock, numThreads>>>();
    }
  }





  //CHECK(hipMemset(d_C, 0, nBytes));

   // gpuRef[0]=1;
   // gpuRef[1]=2;
   // gpuRef[2]=0;
   // CHECK(hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice));
   //
   // testeNum<<<1, 3>>>(numFix,d_C);
   //CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
   // printData("setRowReadRow       ", gpuRef, 3);


  CHECK(hipDeviceReset());
  // hipDeviceReset();

  return 0;
}

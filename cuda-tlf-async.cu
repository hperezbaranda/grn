#include "hip/hip_runtime.h"
#include "common.h"
#include <iostream>
#include <stdio.h>
#include <cstdlib>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define SIZE 3

/*
* A simple introduction to programming in CUDA. This program prints "Hello
* World from GPU! from 10 CUDA threads running on the GPU.
*/
using namespace std;

__device__ bool state[SIZE];
//__shared__ bool equ[SIZE];

__device__ int getDecValue(bool* v){
  int num=0;
  for (size_t i = 0; i < SIZE; i++) {
    //printf("%sd ", (1 << i-1) * v[i-1] );
    num += (1 << i) * v[SIZE-i-1];
  }
  return num;
}

__device__ void initialState(unsigned long int valor, bool *vet, int size) {
  // std::cout <<" initialState: "<< std::endl;
  for (int i = 0; i < size; i++) {
    vet[size-1-i] = (valor & 1) != 0;
    valor >>= 1;
  }
}

__device__ void calculateState(bool* state,int num){
  // for (size_t i = 0; i < SIZE; i++) {
  //   printf("\n%d\n", equ[i]);
  // }
  bool equ[SIZE];
  equ[0] = ( state[0]  | !  state[2] );
  equ[1] = ( state[0]  &   state[2] );
  equ[2] = state[1];
  state[num] = equ[num];

}

__global__ void initSharedMem(){
  for (size_t i = 0; i < SIZE; i++) {
    printf("\n%d\n", state[i]);
  }
  // equ[0]=( state[0]  | !state[2] );
  // equ[1]=( state[0]  &  state[2] );
  // equ[2]=state[1];
}

__global__ void findAttractor(int number)
{

  bool sta[SIZE];
  uint thread = blockDim.x * blockIdx.x + threadIdx.x;
  initialState(thread,sta,SIZE);

  printf("Hello World from GPU! number: %d\n", thread);
  printf("My number: %d\n", number);

  calculateState(sta,number);

//  printf("saida: " );
  printf("%d\n", getDecValue(sta));
}

int main(int argc, char **argv)
{
  uint seed = (uint) time(NULL);
  srand(seed);
  uint suffle = rand() %SIZE;
  // std::cout <<   << std::endl;
  size_t numThreads =2;
  //initSharedMem<<<1,1>>>();
  findAttractor<<<1, numThreads>>>(suffle);
  cout << endl;
  // CHECK(hipFree(d_equation));
  CHECK(hipDeviceReset());
  // hipDeviceReset();

  return 0;
}

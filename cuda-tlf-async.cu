#include "hip/hip_runtime.h"
#include "common.h"
#include <iostream>
#include <stdio.h>
#include <cstdlib>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define SIZE 3

/*
* A simple introduction to programming in CUDA. This program prints "Hello
* World from GPU! from 10 CUDA threads running on the GPU.
*/
using namespace std;


void printData(const char *msg, int *in,  const int size)
{
    printf("%s: ", msg);

    for (int i = 0; i < size; i++)
    {
        printf("%5d", in[i]);
        fflush(stdout);
    }

    printf("\n");
    return;
}

__device__ bool state[SIZE];
//__shared__ bool equ[SIZE];

__device__ int getDecValue(bool* v){
  int num=0;
  for (size_t i = 0; i < SIZE; i++) {
    //printf("%sd ", (1 << i-1) * v[i-1] );
    num += (1 << i) * v[SIZE-i-1];
  }
  return num;
}

__device__ void initialState(unsigned long int valor, bool *vet, int size) {
  // std::cout <<" initialState: "<< std::endl;
  for (int i = 0; i < size; i++) {
    vet[size-1-i] = (valor & 1) != 0;
    valor >>= 1;
  }
}

__device__ void calculateState(bool* state,int num){
  // for (size_t i = 0; i < SIZE; i++) {
  //   printf("\n%d\n", equ[i]);
  // }
  bool equ[SIZE];
  equ[0] = ( state[0]  | !  state[2] );
  equ[1] = ( state[0]  &   state[2] );
  equ[2] = state[1];
  state[num] = equ[num];

}

__global__ void initSharedMem(){
  for (size_t i = 0; i < SIZE; i++) {
    printf("\n%d\n", state[i]);
  }
  // equ[0]=( state[0]  | !state[2] );
  // equ[1]=( state[0]  &  state[2] );
  // equ[2]=state[1];
}

__global__ void findAttractor(int number)
{

  bool state[SIZE];
  uint idx = blockDim.x * blockIdx.x + threadIdx.x;
  initialState(idx,state,SIZE);

  printf("Hello World from GPU! number: %d\n", idx);
  printf("My number: %d\n", number);
  hiprandState_t a_number;
  hiprand_init(idx+clock(), 0, 3,  &a_number);
  unsigned int aleatory = hiprand(&a_number);
  printf("aleatory %d\n", aleatory%SIZE );

  calculateState(state,number);
  // __syncthreads();
//  printf("saida: " );
  printf("%d\n", getDecValue(state));
}

__global__ void setRowReadRow(int *out)
{
    // static shared memory
    __shared__ int tile[3][3];

    // mapping from thread index to global memory index
    // unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;;

    // shared memory store operation
    tile[threadIdx.y][threadIdx.x] = idx;

    // wait for all threads to complete
    // __syncthreads();

    // shared memory load operation
    out[idx] = tile[threadIdx.y][threadIdx.x];
}

int main(int argc, char **argv)
{
  uint seed = (uint) time(NULL);
  srand(seed);
  uint suffle = rand() %SIZE;
  // std::cout <<   << std::endl;
  size_t numSimu = 1 << 20;
  size_t numState = 1 << 3;
  size_t numBlock =0;
  if (numState > 1024){
    numBlock = numState/1024;
  }else{
    numBlock = numState;
  }


  size_t numThreads =1024;
  //initSharedMem<<<1,1>>>();
  findAttractor<<<numBlock, numThreads>>>(suffle);

  // int thread = 64;
  // int block = 20;
  // size_t nBytes = thread * sizeof(int)*block;
  // int *d_C;
  // CHECK(hipMalloc((int**)&d_C, nBytes));
  // int *gpuRef  = (int *)malloc(nBytes);
  //
  // CHECK(hipMemset(d_C, 0, nBytes));
  // setRowReadRow<<<block, thread>>>(d_C);
  // CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
  //
  // printData("set row read row   ", gpuRef, thread*block);


  cout << endl;
  // CHECK(hipFree(d_equation));
  CHECK(hipDeviceReset());
  // hipDeviceReset();

  return 0;
}

#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <stdio.h>
#include "./common.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <assert.h>
#include <fstream>
#include <string>
#include <omp.h>

using namespace std;

#define TABLE_SIZE 2048
#define BUCKET_SIZE 200
#define TAMANHO_VETOR 3
#define PESOS_GPU 200

struct HashTable
{
    unsigned long int *atratores;
    unsigned long int *count;
    int period;
};

HashTable * junta_atratores(HashTable &tabela_atratores,  int *pesos,  int *posIni,  int*eqSize,  int *T,  int nEq)
{
    HashTable * resultado;
    size_t nBytes = TABLE_SIZE*(sizeof(HashTable));
    resultado = (HashTable *)malloc(nBytes);
 

    unsigned long int MAX_ESTADO = (unsigned long int)pow(2,nEq);

    //aloca memoria para o resultado
    for(int i = 0; i < TABLE_SIZE; i++)
    {
        resultado[i].atratores = (unsigned long int *)malloc(sizeof(unsigned long int) * BUCKET_SIZE);
        resultado[i].count = (unsigned long int *)malloc(sizeof(unsigned long int));
        resultado[i].count[0] = 0;
        resultado[i].period = 0;
    }

    int var,peso;
    //garantindo o ciclo(alguns estados calculados pela GPU estao fora do ciclo, porque eu nao sei)
    for(int i = 0; i < TABLE_SIZE; i++ )
    {
        if(tabela_atratores.count[i] != 0)
        {
            int hash = 0;
            //pega os dados dos estado
            unsigned long int s0 = tabela_atratores.atratores[i], s1 = tabela_atratores.atratores[i];
            unsigned long int numEstados  = tabela_atratores.count[i];
            //zera a posição da tabela
            tabela_atratores.atratores[i] = 0;
            tabela_atratores.count[i] = 0; 

            //garantir o ciclo
            do
            {
                unsigned long int newEstado = 0;

                for(int j = 0; j < nEq; j++)
                {
                    int cal_new = nEq-1-j;
                    int repre_var = 0;
                    int sum_prod = 0, pos = posIni[j] , eqsize = eqSize[j], Teq = T[j] ;
                    //aplicando a tlf
                    for(int z = 0; z < eqsize;  z++, pos += 2)
                    {
                        var = pesos[pos];
                        repre_var = (nEq-1)-var;
                        peso = pesos[pos+1];
                        sum_prod += ((s0>>repre_var)%2)*peso;
                    }
                    
                    newEstado |= (sum_prod >= Teq) << cal_new;
                } 
                //extrai primeiro atrator do ciclo
                s0 = newEstado;

                newEstado = 0;
                for(int j = 0; j < nEq; j++)
                {
                    int cal_new = nEq-1-j;
                    int repre_var = 0;
                    int sum_prod = 0, pos = posIni[j] , eqsize = eqSize[j], Teq = T[j] ;
                    //aplicando a tlf
                    for(int z = 0; z < eqsize;  z++, pos += 2)
                    {
                        var = pesos[pos];
                        repre_var = (nEq-1)-var;
                        peso = pesos[pos+1];
                        sum_prod += ((s1>>repre_var)%2)*peso;
                    }
                    
                    newEstado |= (sum_prod >= Teq) << cal_new;
                } 
                //extrai primeiro atrator do ciclo
                s1 = newEstado;

                newEstado = 0;
                for(int j = 0; j < nEq; j++)
                {
                    int cal_new = nEq-1-j;
                    int repre_var = 0;
                    int sum_prod = 0, pos = posIni[j] , eqsize = eqSize[j], Teq = T[j] ;
                    //aplicando a tlf
                    for(int z = 0; z < eqsize;  z++, pos += 2)
                    {
                        var = pesos[pos];
                        repre_var = (nEq-1)-var;
                        peso = pesos[pos+1];
                        sum_prod += ((s1>>repre_var)%2)*peso;
                    }
                    
                    newEstado |= (sum_prod >= Teq) << cal_new;
                } 
                //extrai primeiro atrator do ciclo
                s1 = newEstado;


            }while(s0 != s1);

            //calcula o hash do estado
            unsigned long int auxEstado = s0;
            int upperBit = -1, lowerBit = -1;
            hash = 0;
            for(int j = 0; j < nEq; j++)
            {
                if(lowerBit == -1 && (auxEstado%2 == 1))
                    lowerBit = j+1;
                
                if(auxEstado%2 == 1)
                {
                    upperBit = j + 1;
                    hash += upperBit;
                }
                auxEstado=auxEstado>>1;
            }
            hash += (upperBit - lowerBit);

            //procura um balde vazio desde que o estado encontrado nao seja igual ao dos baldes encontrados no caminho
            if(hash < TABLE_SIZE)    
                while(hash < TABLE_SIZE)
                {
                    if(tabela_atratores.atratores[hash] == s0 || tabela_atratores.count[hash] == 0) break;
                    hash++;
                }
            else
            {
                hash = 0;
                while(hash < TABLE_SIZE)
                {
                    if(tabela_atratores.atratores[hash] == s0 || tabela_atratores.count[hash] == 0) break;
                    hash++;
                }
            }
            
            tabela_atratores.atratores[hash] = s0;
            // tabela_atratores.count[hash] = (numEstados >= tabela_atratores.count[hash]) ? numEstados : tabela_atratores.count[hash];
            tabela_atratores.count[hash] +=numEstados;
        }

    }
  

    for(int i = 0; i < TABLE_SIZE; i++)
    {
        if(tabela_atratores.count[i] != 0)
        {
            //extrai os dados do atrator atual
            unsigned long int estado = tabela_atratores.atratores[i];
            unsigned long int numEstados  = tabela_atratores.count[i];

            unsigned long int aux = 0;

            //da um passo com estado para garantir o ciclo
            unsigned long int newEstado = 0; 

            int k = 0;
            resultado[i].atratores[k++] = estado;
            resultado[i].period++;
            tabela_atratores.count[i] = 0; // já contamos o atrator inicial 
            
            //aplica um passo com aux
            for(int j = 0; j < nEq; j++)
            {
                int cal_new = nEq-1-j;
                int repre_var = 0;
                int sum_prod =0, pos = posIni[j] , eqsize = eqSize[j], Teq = T[j] ;
                //aplicando a tlf
                for(int z = 0; z < eqsize;  z++, pos += 2)
                {
                    var = pesos[pos];
                    repre_var = (nEq-1)-var;
                    peso = pesos[pos+1];
                    sum_prod += ((estado>>repre_var)%2)*peso;
                }
                
                aux |= (sum_prod >= Teq) << cal_new;
            }

            //enquanto aux for diferente acha todos os estados do ciclo e armazena o atrator na tabela de resultado
            while(aux != estado)
            {
                //calcula o hash do estado atual
                unsigned long int auxEstado = estado;
                int upperBit = -1, lowerBit = -1, hash = 0;
                for(int j = 0; j < nEq; j++)
                {
                    if(lowerBit == -1 && (auxEstado%2 == 1))
                        lowerBit = j+1;
                    
                    if(auxEstado%2 == 1)
                    {
                        upperBit = j + 1;
                        hash += upperBit;
                    }
                    auxEstado=auxEstado>>1;
                }
                hash += (upperBit - lowerBit);

                //procurando se o estado encontrado no ciclo esta na tabela
                if(hash < TABLE_SIZE) //se o estado estiver na tabela extraimos seus dados
                {
                    while(hash < TABLE_SIZE)
                    {
                        if(tabela_atratores.atratores[hash] == aux) break;
                        hash++;
                    } 
                }
                else
                {
                    hash = 0;
                    while(hash < TABLE_SIZE)
                    {
                        if(tabela_atratores.atratores[hash] == estado) break;
                        hash++;
                    }
                }

                resultado[i].atratores[k++] = aux;
                resultado[i].period++;

                if(hash < TABLE_SIZE) //se o estado estiver na tabela extraimos seus dados
                {
                    numEstados += tabela_atratores.count[hash];
                    tabela_atratores.count[hash] = 0; // já contamos esse estado em algum atrator 
                }
                

                //da um passo com aux
                newEstado = 0; 
                for(int j = 0; j < nEq; j++)
                {
                    int cal_new = nEq-1-j;
                    int repre_var = 0;
                    int sum_prod = 0, pos = posIni[j] , eqsize = eqSize[j], Teq = T[j] ;
                    //aplicando a tlf
                    for(int z = 0; z < eqsize;  z++, pos += 2)
                    {
                        var = pesos[pos];
                        repre_var = (nEq-1)-var;
                        peso = pesos[pos+1];
                        sum_prod += ((aux>>repre_var)%2)*peso;
                    }
                    
                    newEstado |= (sum_prod >= Teq) << cal_new;
                }
                
                aux = newEstado;
                //printf("aux : %lu\n",aux);
            }

            

            resultado[i].count[0] = numEstados;
        }
    }

    return resultado;   
};


//versao CPU
void sincrono_TabelaCPU(const int * pesos, const int *posIni, const int*eqSize, const int *T,const int nEq, HashTable &tabela_atratores, const unsigned long long MIN_ESTADO, const unsigned long long MAX_ESTADO)
{
    #pragma omp parallel private(tabela_atratores,s0,s1,posIni,pesos,eqSize,T)
    #pragma omp for schedule(static)
    for(unsigned long long estado = MIN_ESTADO; estado < MAX_ESTADO; estado++)
    {  
        unsigned long long s0 = estado, s1 = estado;
        int var,peso;
        // cout << "ESTADO: "<<estado <<endl<<endl;
        do
        {
            //da um passo com s0
            unsigned long long newEstado = 0; 
            for(int j = 0; j < nEq; j++)
            {   
                int cal_new = nEq-1-j;
                int repre_var = 0;
                int sum_prod = 0, pos = posIni[j] , eqsize = eqSize[j], Teq = T[j] ;
                //aplicando a tlf
                for(int z = 0; z < eqsize;  z++, pos += 2)
                {
                    var = pesos[pos];
                    repre_var = (nEq-1)-var;
                    peso = pesos[pos+1];
                    // cout << "Var--Peso "<<var<<" "<<peso<< endl;
                    // cout<< "Rep "<< repre_var<<endl;
                    sum_prod += ((s0>>repre_var)%2)*peso;
                }
                    
                
                newEstado |= (sum_prod >= Teq) << cal_new;
                // cout << "New Estado tmp "<<newEstado<<endl;
            }
            
            s0 = newEstado;
            // cout << "ESTADO FINAL "<<s0<<endl;

            //da dois apssos com s1
            newEstado = 0; 
            for(int j = 0; j < nEq; j++)
            {
                int cal_new = nEq-1-j;
                int repre_var = 0;
                int sum_prod = 0, pos = posIni[j] , eqsize = eqSize[j], Teq = T[j] ;
                //aplicando a tlf
                for(int z = 0; z < eqsize;  z++, pos += 2)
                {
                    var = pesos[pos];
                    repre_var = (nEq-1)-var;
                    peso = pesos[pos+1];
                    sum_prod += ((s1>>repre_var)%2)*peso;
                }
                
                newEstado |= (sum_prod >= Teq) << cal_new;
            }
            
            s1 = newEstado;

            newEstado = 0; 
            for(int j = 0; j < nEq; j++)
            {
                int cal_new = nEq-1-j;
                int repre_var = 0;
                int sum_prod = 0, pos = posIni[j] , eqsize = eqSize[j], Teq = T[j] ;
                //aplicando a tlf
                for(int z = 0; z < eqsize;  z++, pos += 2)
                {
                    var = pesos[pos];
                    repre_var = (nEq-1)-var;
                    peso = pesos[pos+1];
                    sum_prod += ((s1>>repre_var)%2)*peso;
                }
                
                newEstado |= (sum_prod >= Teq) << cal_new;
            }
            
            s1 = newEstado;

        }while(s0 != s1);

        //achou o estado em que s0 e s1 se encontram, salva o estado
        //Neste ponto s1 = s0

        //variaveis auxiliares
        unsigned long long estadoAtr = 0;
        int upperBit = -1, lowerBit = -1, hash = 0;
        unsigned long long auxEstado = 0;

        //extrai o estado
        estadoAtr = s1;
        //printf("%lu\n",s0);

        //calcula o hash do estado
        upperBit = 0; lowerBit = 0; hash = 0;
        auxEstado = estadoAtr;
        #pragma unroll
        for(int i = 0; i < nEq; i++)
        {
            if(lowerBit == -1 && (auxEstado%2 == 1))
                lowerBit = i+1;
            
            if(auxEstado%2 == 1)
            {
                upperBit = i + 1;
                hash += upperBit;
            }
            auxEstado=auxEstado>>1;
        }
        hash += (upperBit - lowerBit);
        
        //insere o estado na tabela hash :
        if(hash >= TABLE_SIZE || hash < 0){
            printf("Estado : %llu Erro ao calcular o hash : %d\n",estadoAtr,hash);
            return;
        }

        //confere se o balde já está cheio e acha um balde vazio
        if(tabela_atratores.atratores[hash] != 0 && tabela_atratores.count[hash]  == estadoAtr)
            tabela_atratores.count[hash]++;//se dois estados caem no mesmo balde, soma mais um no estado
        else
        {
            //procura um balde vazio desde que o estado encontrado nao seja igual ao dos baldes encontrados no caminho
            while(tabela_atratores.count[hash] != 0 && tabela_atratores.atratores[hash] != estadoAtr) hash++;

            if(hash >= TABLE_SIZE) hash = 13;
            tabela_atratores.atratores[hash] = estadoAtr;
            tabela_atratores.count[hash]++;
        }
    }
}



//versao GPU
__device__ __constant__ int pesosGPU[PESOS_GPU];

__global__ void sincrono_Tabela(const int *posIni ,const int*eqSize,const int *T, unsigned long int *atratores, unsigned long int *count ,const int nEq,const unsigned long long MIN_ESTADO,  const unsigned long long MAX_ESTADO)
{
    //idx da thread também será o estado do grafo
    unsigned long int idx = blockDim.x*blockIdx.x + threadIdx.x + MIN_ESTADO;

    if(idx < MAX_ESTADO)
    {
        //definicao dos vetores em shared memory
        __shared__ int sh_posIni[TAMANHO_VETOR];
        __shared__ int sh_eqSize[TAMANHO_VETOR];
        __shared__ int sh_T[TAMANHO_VETOR];

        __shared__ unsigned long long stable[2][TABLE_SIZE];

        for(unsigned int j = 0; j < (TABLE_SIZE/blockDim.x); j++)
        {
            stable[0][threadIdx.x*(TABLE_SIZE/blockDim.x)+j] = 0;
            stable[1][threadIdx.x*(TABLE_SIZE/blockDim.x)+j] = 0;
        }

        if(threadIdx.x < nEq)
        {
            sh_posIni[threadIdx.x] = posIni[threadIdx.x];
            sh_eqSize[threadIdx.x] = eqSize[threadIdx.x];
            sh_T[threadIdx.x] = T[threadIdx.x];
        }
            
        __syncthreads();
    
        //cada thread faz uma simulação por vez
        unsigned long int s0=idx, s1=idx;

        unsigned long int aux = 0, newEstado = 0;
        
        do
        {
            //guarda o estado em aux
            aux = s0;
            newEstado = 0;

            //um passo em s0
            for(int i = 0; i < nEq; i++)
            {
                int cal_new = nEq-1-i;
                int repre_var = 0;
                int sum_prod =0, pos = sh_posIni[i] , eqsize = sh_eqSize[i], Teq = sh_T[i] ;
                //aplicando a tlf
                for(int j = 0; j < eqsize;  j++, pos += 2){
                    repre_var = (nEq-1)-pesosGPU[pos];
                    sum_prod += ((aux>>repre_var)%2)*pesosGPU[pos+1];
                }
                
                newEstado |= (sum_prod >= Teq) << cal_new;
            }
            s0 = newEstado;
            
            //dois passos em s1
            aux = s1;
            newEstado = 0;

            for(int i = 0; i < nEq; i++)
            {
                int cal_new = nEq-1-i;
                int repre_var = 0;
                int sum_prod =0, pos = sh_posIni[i] , eqsize = sh_eqSize[i], Teq= sh_T[i] ;
                //aplicando a tlf
                for(int j = 0; j < eqsize;  j++, pos += 2){
                    repre_var = (nEq-1)-pesosGPU[pos];
                    sum_prod += ((aux>>repre_var)%2)*pesosGPU[pos+1];
                }
                
                newEstado |= (sum_prod >= Teq) << cal_new;
            }

            aux = newEstado;
            newEstado = 0;
            for(int i = 0; i < nEq; i++)
            {
                int cal_new = nEq-1-i;
                int repre_var = 0;
                int sum_prod =0, pos = sh_posIni[i] , eqsize = sh_eqSize[i], Teq= sh_T[i] ;
                //aplicando a tlf
                for(int j = 0; j < eqsize;  j++, pos += 2){
                    repre_var = (nEq-1)-pesosGPU[pos];
                    sum_prod += ((aux>>repre_var)%2)*pesosGPU[pos+1];
                }
                
                newEstado |= (sum_prod >= Teq) << cal_new;
            }

            s1 = newEstado;

        }while(s0 != s1);

        __syncthreads();

        //Neste ponto s1 = s0

        //variaveis auxiliares
        unsigned long int estado = 0;
        int upperBit = -1, lowerBit = -1, hash = 0;
        unsigned long int auxEstado = 0;

        //extrai o estado
        estado = s1;
        //printf("%lu\n",s0);

        //calcula o hash do estado
        upperBit = 0; lowerBit = 0; hash = 0;
        auxEstado = estado;
        #pragma unroll
        for(int i = 0; i < nEq; i++)
        {
            if(lowerBit == -1 && (auxEstado%2 == 1))
                lowerBit = i+1;
            
            if(auxEstado%2 == 1)
            {
                upperBit = i + 1;
                hash += upperBit;
            }
            auxEstado=auxEstado>>1;
        }
        hash += (upperBit - lowerBit);
        
        //insere o estado na tabela hash :
        if(hash >= TABLE_SIZE || hash < 0){
            printf("Estado : %lu Erro ao calcular o hash : %d\n",estado,hash);
            return;
        }

        //confere se o balde já está cheio e acha um balde vazio
        if(stable[1][hash] != 0 && stable[0][hash] == estado)
            atomicAdd((unsigned long long *)&(stable[1][hash]),(unsigned long long)1);//se dois estados caem no mesmo balde, soma mais um no estado
        else
        {
            //procura um balde vazio desde que o estado encontrado nao seja igual ao dos baldes encontrados no caminho
            while(stable[1][hash] != 0 && stable[0][hash] != estado) hash++;

            if(hash >= TABLE_SIZE) hash = 13;
            atomicExch((unsigned long long *)&(stable[0][hash]),(unsigned long long)estado);
            atomicAdd((unsigned long long *)&(stable[1][hash]),(unsigned long long)1); 
        }
        __syncthreads();

        if(threadIdx.x == 0)
        {
            #pragma unroll
            for(int i = 0; i < TABLE_SIZE; i++)
            {
                atomicAdd((unsigned long long *)&(atratores[i]), (unsigned long long)stable[0][i]);
                atomicAdd((unsigned long long *)&(count[i]), (unsigned long long)stable[1][i]);
            } 
        }
        __syncthreads();

    }

}

/* __global__ void assincrono_Tabela(hiprandState * curandstate, const int *posIni ,const int*eqSize,const int *T, unsigned long long * result ,const int nEq,const unsigned long long MIN_ESTADO,  const unsigned long long MAX_ESTADO)
{

    int idx = threadIdx.x+blockDim.x*blockIdx.x;

    //definindo qual equação a thread pega
    int warpID = threadIdx.x / 32;
    int nWarps = blockDim.x / 32;

    int EQ = blockIdx.x*nWarps + warpID;

    //inicia o rand usando o id da tread + estado min como seed
    hiprand_init(idx+MIN_ESTADO,MAX_ESTADO - MIN_ESTADO,MIN_ESTADO,&curandstate[idx]);

    //so faz a computacao se a equacao carregada é valida
    if(EQ >= 0 && EQ < nEq)
    {
        //printf("Block : %d Warp : %d Thread : %d  EQ : %d\n",blockIdx.x,warpID,threadIdx.x,EQ);
        int pos = posIni[EQ], eqsize = eqSize[EQ], Teq = T[EQ];

        //definindo o numero de simulacoes e valor inicial do contador de quantas vezes a variavel e zero
        unsigned long long nSim = 1000000/32, zero = 0;

        for(unsigned long long i = 0; i < nSim; i++)
        {
            //gerando estado aleatório a partir de uma distribuição uniforme
            float randf = hiprand_uniform(&(curandstate[idx]));
            randf *= (MAX_ESTADO - MIN_ESTADO  + 0.999999);
            randf += MIN_ESTADO;
            unsigned long long estado = (unsigned long long)truncf(randf);

            int sum_prod =0;
            //aplicando a tlf
            for(int j = 0; j < eqsize;  j++, pos += 2)
                sum_prod += ((estado>>pesosGPU[pos])%2)*pesosGPU[pos+1];
            //se a soma for menor que o treshold o valor da variável é zero
            zero += (sum_prod < Teq) ? 1 : 0;
        }
        __syncthreads();

        //shuffle_xor
        //traz todos os dados para a tread 0 de cada warp e salva no vetor
        for(int mask = 1; mask <= 16; mask*=2 )
            zero += __shfl_xor (zero, mask,32);
        __syncthreads();
        

        int laneID = threadIdx.x%32;
        //retorna os resultados para a CPU
        if(laneID == 0)
            atomicAdd(&result[EQ],zero);
        __syncthreads();

    }

} */


int main(int argc, char **argv)
{
    int nEq; // numero de equações
    filebuf fb;
    if(!fb.open(argv[1],ios::in))
    {
        cerr << "Erro ao abrir arquivo de entrada " << argv[1] <<endl;
        exit(0);
    }

    istream is(&fb);
    is >> nEq;

    int * pesosCPU, *pesoIniCPU, *pesoIniGPU, *eqSizeCPU, *eqSizeGPU,*TCPU,*TGPU;
    unsigned long long *resultCPU, *resultGPU; // peso, tamanho das equações e threshold de cada equação
    size_t bytes = sizeof(int)*nEq;

    // Alocando memória do host
    eqSizeCPU = (int *)malloc(bytes);
    TCPU = (int *)malloc(bytes);
    pesoIniCPU = (int *)malloc(bytes);
    resultCPU = (unsigned long long *)malloc(3*sizeof(unsigned long long));

    //alocando memoria device
    hipMalloc((int **)&eqSizeGPU,bytes);
    hipMalloc((int**)&TGPU,bytes);
    hipMalloc((int **)&pesoIniGPU, bytes);
    hipMalloc((unsigned long long **)&resultGPU, 3*sizeof(unsigned long long));

    //lendo os tamanhos das equações
    int nPesos = 0; // numero de pesos

    for(int i = 0; i < 3; i++)
        resultCPU[i] = 0;

    for(int i = 0; i < nEq; i++)
    {
        is >> eqSizeCPU[i];
        nPesos+= eqSizeCPU[i];
    }
    
    //alocando vetores com pesos e re
    pesosCPU = (int *)malloc(sizeof(int)*nPesos*2);

    int posPeso = 0; //posição dos pesos
    for(int i = 0; i < nEq; i++)
    {
        int var=0, peso=0, T=0; // variavel da equação, peso e threshold
        pesoIniCPU[i] = posPeso;
        for(int j = 0; j < eqSizeCPU[i]; j++,posPeso+=2)
        {
            is >> var >> peso;
            pesosCPU[posPeso] = var;
            pesosCPU[posPeso + 1] = peso;
        }
        is >> T;
        TCPU[i] = T;
    }
    fb.close();

    
     //pesos ficam em memoria const
    hipMemcpy(TGPU,TCPU,bytes,hipMemcpyHostToDevice);
    hipMemcpy(eqSizeGPU,eqSizeCPU,bytes,hipMemcpyHostToDevice);
    hipMemcpy(pesoIniGPU,pesoIniCPU,bytes,hipMemcpyHostToDevice);
    hipMemcpy(resultGPU,resultCPU,3*sizeof(unsigned long long),hipMemcpyHostToDevice);

    // ------------------- Sincrono -----------------

     size_t nBytes = sizeof(unsigned long int)*TABLE_SIZE;

    //aloca memoria 
    HashTable cpuTable,gpuTable, cpuTable2;
    cpuTable.atratores = (unsigned long int *)malloc(nBytes);
    cpuTable.count = (unsigned long int *)malloc(nBytes);
    cpuTable2.atratores = (unsigned long int *)malloc(nBytes);
    cpuTable2.count = (unsigned long int *)malloc(nBytes);
    hipMalloc((unsigned long int **)&gpuTable.atratores, nBytes);
    hipMalloc((unsigned long int **)&gpuTable.count,nBytes);

    //inicializacao da tabela
    for(unsigned int i = 0; i < TABLE_SIZE; i++)
    {
        cpuTable.count[i] = 0;
        cpuTable.atratores[i] = 0;
    }
    
    //copia tabela do host para tabela da gpu
    hipMemcpy(gpuTable.atratores, cpuTable.atratores, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(gpuTable.count, cpuTable.count, nBytes, hipMemcpyHostToDevice);

    int threads = 1024;
    dim3 block(threads);

    unsigned long long MIN_ESTADO = 0;
    unsigned long long MAX_ESTADO = 0;

    //pede a quantidade de estados para simular
    string argv2 = argv[2];
    for(int i = 0; i < argv2.size() ; i++)
        MAX_ESTADO += ((unsigned long int)(argv2[i] - '0'))*pow(10,argv2.size()-i-1);
    
    dim3 grid((MAX_ESTADO + block.x -1)/block.x);

    //GPU
    /* printf("Modelo Sincrono GPU\n");
    printf("N de estados : %llu\n",MAX_ESTADO - MIN_ESTADO);
    printf("Blocos : %d  Threads : %d \n",grid.x, block.x); */
    HashTable * resultado;
    string tec = argv[3];
    
    if(tec == "GPU")
    {
        hipMemcpyToSymbol(HIP_SYMBOL(pesosGPU),pesosCPU,sizeof(int)*nPesos*2);//copia memoria do host para o device
        sincrono_Tabela<<<grid,block,bytes>>>(pesoIniGPU,eqSizeGPU,TGPU,gpuTable.atratores, gpuTable.count,nEq,MIN_ESTADO,MAX_ESTADO);
           
        //extrai resultado da GPU
        hipMemcpy(cpuTable.atratores, gpuTable.atratores, nBytes, hipMemcpyDeviceToHost);
        hipMemcpy(cpuTable.count, gpuTable.count, nBytes, hipMemcpyDeviceToHost);

        //estados de todos os atratores separados
        /* for(int i = 0; i < TABLE_SIZE; i++)
            printf("%lu %lu\n",cpuTable.atratores[i], cpuTable.count[i]); */

        //Faz o estograma (HashTable &tabela_atratores, const int *pesos, const int *posIni, const int*eqSize, const int *T, const int nEq)
        resultado = junta_atratores(cpuTable,pesosCPU,pesoIniCPU,eqSizeCPU,TCPU,nEq);
        //imprime resultado
        for(int i = 0; i < TABLE_SIZE; i++)
        {
            if(resultado[i].period != 0 )
            {
                cout<<resultado[i].period<< " ";
                for(int j = 0; j < resultado[i].period; j++)
                    cout<<resultado[i].atratores[j]<< " ";
                cout<<resultado[i].count[0]<<endl;
            }
        }

        //libera memoria alocada
        for(int i = 0; i < TABLE_SIZE; i++)
        {
            free(resultado[i].atratores);
            free(resultado[i].count);
        }
        free(resultado);
        hipFree(gpuTable.atratores);
        hipFree(gpuTable.count);
    }

    
    HashTable * resultadoCPU;

    if(tec == "CPU")
    {
        /* printf("Modelo Sincrono CPU\n"); */
        //CPU

        
        sincrono_TabelaCPU(pesosCPU,pesoIniCPU,eqSizeCPU,TCPU,nEq,cpuTable2,MIN_ESTADO,MAX_ESTADO);
        resultadoCPU = junta_atratores(cpuTable2,pesosCPU,pesoIniCPU,eqSizeCPU,TCPU,nEq);
        
        //imprime resultado
        for(int i = 0; i < TABLE_SIZE; i++)
        {
            if(resultadoCPU[i].period != 0 )
            {
                cout << resultadoCPU[i].period<< " ";
                for(int j = 0; j < resultadoCPU[i].period; j++)
                    cout << resultadoCPU[i].atratores[j]<<" ";
                cout<<resultadoCPU[i].count[0]<<endl;
            }
        }

        //libera memoria alocada
        for(int i = 0; i < TABLE_SIZE; i++)
        {
            free(resultadoCPU[i].atratores);
            free(resultadoCPU[i].count);
        }
    

        
        free(resultadoCPU);
        free(cpuTable.atratores);
        free(cpuTable.count);
    }

    //-------------Assincrono----------------

    /* hiprandState *d_state;
    hipMalloc(&d_state, sizeof(hiprandState));

    //limpa a memoria do resultado para reutilizar o vetor
    free(resultCPU);
    hipFree(resultGPU);

    size_t bytesResult = nEq*sizeof(unsigned long long);

    //aloca vetor de resultado
    resultCPU = (unsigned long long *)malloc(bytesResult);
    hipMalloc((unsigned long long **)&resultGPU, bytesResult);

    //inicializa valores
    for(int i = 0; i < nEq ; i++)
        resultCPU[i] = 0;
    
    hipMemcpy(resultGPU, resultCPU, bytesResult, hipMemcpyHostToDevice);


    printf("Modelo Assincrono\n");
    printf("Estado Min : %llu Estado Max : %llu\n",MIN_ESTADO,MAX_ESTADO);
    printf("Blocos : %d  Threads : %d \n",grid.x, block.x);
    assincrono_Tabela<<<grid,block,bytes>>>(d_state,pesoIniGPU,eqSizeGPU,TGPU,resultGPU,nEq,MIN_ESTADO,MAX_ESTADO);
    CHECK(hipGetLastError());

    hipMemcpy(resultCPU,resultGPU,bytesResult,hipMemcpyDeviceToHost);

    double async_p0[nEq];

    for(int i = 0; i < nEq; i++)
    {
        async_p0[i] = ((double)resultCPU[i]/(double)(MAX_ESTADO-MIN_ESTADO));
        printf("Variavel  %d  0: %2f  1: %2f\n",i,async_p0[i],1.000000000-async_p0[i]);
    } */
    
    free(TCPU);
    free(eqSizeCPU);
    free(pesosCPU);
    free(pesoIniCPU);
    hipFree(eqSizeGPU);
    hipFree(TGPU);
    hipFree(pesoIniGPU);
    
    
    hipDeviceReset();
    return 0;
}

